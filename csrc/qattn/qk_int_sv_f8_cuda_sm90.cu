#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by SageAttention team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../utils.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/extension.h>

#include "../wgmma.cuh"
#include "../math.cuh"
#include "../dispatch_utils.h"

#include "attn_utils.cuh"

// <NT> ����һ�� 4 ά����ӳ�䣬ͨ�� cuTensorMapEncodeTiled ��ȫ���ڴ��е�����ӳ�䵽�����ڴ��еĿ顣
// ���ڸ�Ч�ع�����������ά���飩���ڴ���ʣ��ڴ沼���� d1-d4 [batch_size, nhead, seq_len, head_dim]. gmem_prob_shapep[head_dim, seq_len, nhead, batch_size]
template <int BlockMajorSize, int BlockMinorSize, bool swizzle=true, CUtensorMapL2promotion_enum promotion_mode=CU_TENSOR_MAP_L2_PROMOTION_NONE, typename T>
CUtensorMap create_tensor_map_4D(T* gmem_ptr, int d1, int d2, int d3, int d4, int stride1, int stride2, int stride3) {
    constexpr int smem_stride = BlockMinorSize * sizeof(T);
    static_assert(sizeof(T) == 2 || sizeof(T) == 1);
    static_assert(smem_stride == 32 || smem_stride == 64 || smem_stride == 128);
    
    CUtensorMap tma_map;
    void* gmem_address = (void*)gmem_ptr;
    uint64_t gmem_prob_shape[5] = {(uint64_t)d4, (uint64_t)d3, (uint64_t)d2, (uint64_t)d1, 1};
    uint64_t gmem_prob_stride[5] = {(uint64_t)stride3 * sizeof(T), (uint64_t)stride2 * sizeof(T), (uint64_t)stride1 * sizeof(T), 0, 0};
    uint32_t smem_box_shape[5] = {uint32_t(BlockMinorSize), uint32_t(BlockMajorSize), 1, 1, 1};
    uint32_t smem_box_stride[5] = {1, 1, 1, 1, 1};

    hipError_t result = cuTensorMapEncodeTiled(
        &tma_map, (sizeof(T) == 2) ? CU_TENSOR_MAP_DATA_TYPE_BFLOAT16 : CU_TENSOR_MAP_DATA_TYPE_UINT8, 4, gmem_address, gmem_prob_shape,
        gmem_prob_stride, smem_box_shape, smem_box_stride, CU_TENSOR_MAP_INTERLEAVE_NONE,
        (swizzle == false) ? CU_TENSOR_MAP_SWIZZLE_NONE : (smem_stride == 128) ? CU_TENSOR_MAP_SWIZZLE_128B : (smem_stride == 64) ? CU_TENSOR_MAP_SWIZZLE_64B : CU_TENSOR_MAP_SWIZZLE_32B, 
        promotion_mode, CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE);

    assert(result == hipSuccess);

    return tma_map;
}

// <NT> __cvta_generic_to_shared��ͨ��ָ��barת��Ϊ�����ڴ��ַ. 
// CUDA���ڴ�ģ���У���ͬ���͵��ڴ�(gmem��smem��)�в�ͬ�ĵ�ַ�ռ�.
__device__ __forceinline__ void init_barrier(uint64_t* bar, int thread_count) {
    uint32_t bar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar)); 
    asm volatile (
        "mbarrier.init.shared::cta.b64 [%0], %1;\n"
        :: "r"(bar_ptr), "r"(thread_count)
    );
}

// <NT> ȷ����ָ���Ĺ����ڴ��ַbar����Ԥ�ڵ��ֽ���bytes�Ѿ�׼����.
// ���ڶ��̻߳��߳̿�֮���Э����ȷ���ڼ���ִ��֮ǰ��������ص����ݶ��Ѿ�д�빲���ڴ档
template <uint32_t bytes>
__device__ __forceinline__ void expect_bytes(uint64_t* bar) {
    uint32_t bar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar)); 
    asm volatile ("mbarrier.arrive.expect_tx.shared::cta.b64 _, [%0], %1;\n"
        :: "r"(bar_ptr), "n"(bytes));
}

// <NT> src_tma_map�Ǻ���create_tensor_map_4D�ķ���ֵ��dst�ǹ����ڴ棬ͨ��map�����ݴ�gmemת��smem
template <typename T>
__device__ __forceinline__ void load_async_4D(T *dst, void const* const src_tma_map, uint64_t* bar, int s0, int s1, int s2, int s3) {
    uint64_t tma_ptr  = reinterpret_cast<uint64_t>(src_tma_map);
    uint32_t mbar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar));
    uint32_t dst_ptr  = static_cast<uint32_t>(__cvta_generic_to_shared(dst));

    asm volatile (
        "cp.async.bulk.tensor.4d.shared::cluster.global.tile.mbarrier::complete_tx::bytes"
        " [%0], [%1, {%3, %4, %5, %6}], [%2];"
        :
        : "r"(dst_ptr), "l"(tma_ptr), "r"(mbar_ptr),
        "r"(s0), "r"(s1), "r"(s2), "r"(s3)
        : "memory"
    );
}

template <typename T>
__device__ __forceinline__ void store_async_4D(void const* dst_tma_map, T *src, int global_token_idx, int global_head_idx, int global_batch_idx) {
    uint64_t tma_ptr  = reinterpret_cast<uint64_t>(dst_tma_map);
    uint32_t src_ptr  = static_cast<uint32_t>(__cvta_generic_to_shared(src));

    asm volatile (
        "cp.async.bulk.tensor.4d.global.shared::cta.tile.bulk_group"
        " [%0, {%2, %3, %4, %5}], [%1];"
        :
        : "l"(tma_ptr), "r"(src_ptr),
        "n"(0), "r"(global_token_idx), "r"(global_head_idx), "r"(global_batch_idx)
        : "memory"
    );
}

// <NT> mbarrier.try_wait.parity.shared::cta.b64 ���Եȴ��ڴ����ϣ�ֱ�������ض�����
// P1 ����洢�ڲ����Ĵ���P1��, @P1 bra.uni DONE���P1Ϊ�棨���ڴ������������㣩������������ת��DONE��ǩ��
// bra.uni LAB_WAIT;�����P1Ϊ�٣����ڴ��������������㣩������������ת��LAB_WAIT��ǩ�������ȴ�
// kPhaseBit ָ���ڴ����ϵĽ׶Σ�������һ��λ���룬����ָ����ǰ�̻߳��߳̿���Ҫ�ȴ����ض��׶Ρ�
// ��wait������expect_bytes����ʹ��, expect_bytes��ʾϣ�������ʹwait�ǵȴ����ϣ����ɡ�
__device__ __forceinline__ void wait(uint64_t* bar, int kPhaseBit) {
    uint32_t mbar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar)); 
    asm volatile (
        "{\n"
        ".reg .pred                P1;\n"
        "LAB_WAIT:\n"
        "mbarrier.try_wait.parity.shared::cta.b64 P1, [%0], %1;\n"
        "@P1                       bra.uni DONE;\n"
        "bra.uni                   LAB_WAIT;\n"
        "DONE:\n"
        "}\n"
        :: "r"(mbar_ptr),
        "r"(kPhaseBit)
    );
}

template <uint32_t count = 1>
__device__ __forceinline__ void arrive(uint64_t* bar) {
    uint32_t mbar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar)); 
    asm volatile (
        "mbarrier.arrive.release.cta.shared::cta.b64 _, [%0], %1;\n"
        :
        : "r"(mbar_ptr), "n"(count)
        : "memory"
    );
}

// <NT> sm90��attention����cuda kernel (��cutlass)��qkΪint8��vΪfp8.
// ������·��sageattn "sm90"�ɷ� 
//          -> sageattn_qk_int8_pv_fp8_cuda_sm90 
//             -> qk_int8_sv_f8_accum_f32_fuse_v_scale_attn_inst_buf 
//                -> qk_int8_sv_f8_attn_kernel
// launch����: grid(div_ceil(qo_len, CTA_Q), num_qo_heads, batch_size)��CTA_Q��һ��block�����Qά��������q�ܳ�����CTA_Q��ʾQά���������block������
//             ����num_qo_heads��batch_size�����һ����ά��grid����block��һά128���̡߳�
// CTA_Q ��ʾһ��block��Ҫ�����q��ά��(�̶�Ϊ64)�� CTA_K��ʾһ��block��Ҫ�����k��ά��(�̶�Ϊ128)��
// Q_GRAN��K_GRANͨ��һ�£�ȡper_warp��per_thread����������������sageattn_qk_int8_pv_fp8_cuda_sm90��
// DTypeOut��bf16��fp16��mask_modeֻȡ�Ƿ������mask��return_lseͨ��Ϊfalse���ɽӿ�sageattn����ú�������fuse_v_scaleΪtrue��
// sm_scale: ��δ�ṩ����ȡhead_dim_og**-0.5����1/����(head_dim), �������ŵ�����Է�ֹ�ڼ��� softmax ʱ��ֵ�����µ���ֵ���ȶ���
//           Χ��head_dim���У�����Ϊ�ȶ�����head_dim��С�йأ�head_dim�ϴ���ô����Ľ�����ܻ�ǳ���
template<uint32_t CTA_Q, uint32_t CTA_K, uint32_t NUM_THREADS, uint32_t head_dim, QuantGranularity Q_GRAN, QuantGranularity K_GRAN, typename DTypeOut, MaskMode mask_mode = MaskMode::kNone, bool return_lse = false, bool fuse_v_scale=false>
__global__ void qk_int8_sv_f8_attn_kernel(const __grid_constant__ CUtensorMap tensorMapQ, 
                                        const __grid_constant__ CUtensorMap tensorMapK,
                                        const __grid_constant__ CUtensorMap tensorMapV,
                                        float *__restrict__ Q_scale, float *__restrict__ K_scale, float *__restrict__ V_scale,
                                        DTypeOut* O, float *__restrict__ Lse, uint32_t stride_bz_o, uint32_t stride_h_o, uint32_t stride_seq_o,
                                        const uint32_t qo_len, const uint32_t kv_len, const uint32_t num_kv_groups,
                                        float sm_scale)
{
  static_assert(NUM_THREADS == 128);
  static_assert(CTA_Q <= CTA_K);
  
  const uint32_t warp_idx = (threadIdx.x % 128) / 32;
  const uint32_t lane_id = threadIdx.x % 32;

  // <NT> һ��block���зֵ�tile��CTA_QΪ64����Qά���ϲ������з֣�CTA_KΪ128��һ��block��kά�Ȼ��8��tile������, head_dimΪ64��128.
  // ������: 
  // num_tiles_q=1,      num_tiles_k=8,         num_tiles_qk_inner=2/4
  // num_tiles_v=4/8,    num_tiles_pv_inner=4
  constexpr uint32_t num_tiles_q = CTA_Q / 64;
  constexpr uint32_t num_tiles_k = CTA_K / 16;
  constexpr uint32_t num_tiles_qk_inner = head_dim / 32;
  constexpr uint32_t num_tiles_v = head_dim / 16;
  constexpr uint32_t num_tiles_pv_inner = CTA_K / 32;

  const uint32_t batch_id = blockIdx.z;
  const uint32_t bx = blockIdx.x;
  const uint32_t head_id = blockIdx.y;
  const uint32_t num_qo_heads = gridDim.y;
  const uint32_t kv_head_id = head_id / num_kv_groups;

  // <NT> ���������Ӵ���Ȼ�����ռ�ת������ 2 Ϊ�׵Ķ����ռ䡣
  // ������ֱ�Ӹ�dequant_scale��ˡ�
  sm_scale *= math::log2e;

  extern __shared__ __align__(128) int8_t smem_[];

  // <NT> sQ[CTA_Q, head_dim], sK[CTA_K, head_dim], sV[head_dim, CTA_K]]
  // sQ*sKt=P[CTA_Q, CTA_K], P*sV=O[CTA_Q, head_dim]
  int8_t *sQ = (int8_t*)smem_;
  int8_t *sK = (int8_t*)(smem_ + CTA_Q * head_dim * sizeof(int8_t));
  int8_t *sV = (int8_t*)(smem_ + CTA_Q * head_dim * sizeof(int8_t) + CTA_K * head_dim * sizeof(int8_t));
  half *sO = (half*)smem_;

  // <NT> ���wgmma��ָ��ֿ飬ָ��ȡm64n128k32 �� m64n128k32����һ��block���ǵ����ݻ�������ָ��tile��
  // ����һ��block�ж��ٸ�ָ��tile������ͷֶ��ٸ�Ԫ�ء�
  // RS������int32_t����Ϊ��QK��mma��s8*s8=s32��ROȡfloat����ΪPV��mma��f8*f8=f32.
  int32_t RS[num_tiles_q][num_tiles_k][8];
  float RO[num_tiles_q][num_tiles_v][8];
  float m[num_tiles_q][2];
  float d[num_tiles_q][2];

  uint32_t q_scale_idx, k_scale_idx;

  if constexpr (Q_GRAN == QuantGranularity::kPerBlock)
  {
    const uint32_t num_block_q = gridDim.x;
    q_scale_idx = batch_id * num_qo_heads * num_block_q + head_id * num_block_q + bx;
  }
  else if constexpr (Q_GRAN == QuantGranularity::kPerWarp)
  {
    const uint32_t num_warp_block_q = gridDim.x * 4;
    q_scale_idx = batch_id * num_qo_heads * num_warp_block_q + head_id * num_warp_block_q + bx * 4 + warp_idx;
  }
  else if constexpr (Q_GRAN == QuantGranularity::kPerThread)
  {
    const uint32_t num_warp_block_q = gridDim.x * 4;
    q_scale_idx = batch_id * num_qo_heads * (num_warp_block_q * 8) + head_id * (num_warp_block_q * 8) + bx * (4 * 8) + warp_idx * 8 + lane_id / 4;
  }

  if constexpr (K_GRAN == QuantGranularity::kPerBlock || K_GRAN == QuantGranularity::kPerWarp)
  {
    const uint32_t num_block_k = div_ceil(kv_len, CTA_K);
    k_scale_idx = batch_id * (num_qo_heads / num_kv_groups) * num_block_k + (head_id / num_kv_groups) * num_block_k;
  }
  else if constexpr (K_GRAN == QuantGranularity::kPerThread)
  {
    const uint32_t num_block_k = div_ceil(kv_len, CTA_K);
    k_scale_idx = batch_id * (num_qo_heads / num_kv_groups) * (num_block_k * 4) + (head_id / num_kv_groups) * (num_block_k * 4) + lane_id % 4;
  }

  constexpr uint32_t k_scale_advance_offset = (K_GRAN == QuantGranularity::kPerBlock || K_GRAN == QuantGranularity::kPerWarp) ? 1 : 4;

  uint32_t Q_idx_lane_base = bx * CTA_Q + warp_idx * 16 + lane_id / 4;

#pragma unroll
  for (uint32_t fq = 0; fq < num_tiles_q; fq++)
  {
    m[fq][0] = -5000000.0f;
    m[fq][1] = -5000000.0f;
    d[fq][0] = 1.0f;
    d[fq][1] = 1.0f;
  }

#pragma unroll
  for (uint32_t fq = 0; fq < num_tiles_q; fq++)
  {
#pragma unroll
    for (uint32_t fv = 0; fv < num_tiles_v; fv++)
    {
#pragma unroll
      for (uint32_t k = 0; k < 8; k++)
      {
        RO[fq][fv][k] = 0.0f;
      }
    }
  }

  __shared__ __align__(8) uint64_t barrier_Q;
  __shared__ __align__(8) uint64_t barrier_K;
  __shared__ __align__(8) uint64_t barrier_V;

  if (threadIdx.x == 0) {
    init_barrier(&barrier_Q, 1);
    init_barrier(&barrier_K, 1);
    init_barrier(&barrier_V, 1);
  }

  __syncthreads();

  // <NT> ͬʱ����qkv��tma�첽����ָ�һ��blockʹ��һ���̷߳���
  // expect_bytes��Ԥ�ڸ�barrier������Ӧ���ֽ������ʹwait���ȴ����ʹ���ɡ�
  // load_async_4D��gmem_prob_shapep[head_dim, seq_len, nhead, batch_size]
  // bx = blockIdx.x;  head_id = blockIdx.y; batch_id = blockIdx.z;  kv_head_idת��head_id
  // 
  // Q��block��xyz�漰����Q��seq_len / nhead / batch_size����һ�λᱻȫ��ȡ�����м��㣬
  // ֻ��head_dimδ�漰������head_dim���ֻ��128, ��һ��block��Χ�ڣ�
  // ����Q�Ķ�ȡ����load_async_4D(sQ, &tensorMapQ, &barrier_Q, 0, bx * CTA_Q, head_id, batch_id);һ�ο�����ȡ����
  // K: block��yz�漰����kv_head_id / batch_id����һά��head_dimͬ��Ҳ��һ��block�ڣ�ʣ��seq_len��ά��δ������ȡ����
  //    �������forѭ�������У���Ҫ���ζ�ȡseq_lenά���ϵĿ飬���зֿ鴦��
  // V����K���ơ�
  // ����һ��block��Ҫ����Q��һ��tile����ѭ��ȡKV��seq_lenά�ȵĶ��tile�ļ��㡣
  // load Q
  // load K0��V0
  // for��
  //    mma(mma(Q,Ki), Vi)
  //    load Ki+1, Vi+1
  //
  // load Q, K, V
  if (threadIdx.x == 0)
  {
    expect_bytes<(CTA_Q * head_dim) * sizeof(int8_t)>(&barrier_Q);
    expect_bytes<(CTA_K * head_dim) * sizeof(int8_t)>(&barrier_K);
    expect_bytes<(CTA_K * head_dim) * sizeof(int8_t)>(&barrier_V);
    load_async_4D(sQ, &tensorMapQ, &barrier_Q, 0, bx * CTA_Q, head_id, batch_id);
    load_async_4D(sK, &tensorMapK, &barrier_K, 0, 0, kv_head_id, batch_id);
    load_async_4D(sV, &tensorMapV, &barrier_V, 0, 0, kv_head_id, batch_id);
  }

  float q_scale = Q_scale[q_scale_idx];
  float original_sm_scale = sm_scale;

  // <NT> wait��sQ�ѵ�λ, ���Կ�ʼforѭ�����KV��ļ��㡣
  // wait for Q
  wait(&barrier_Q, 0);

  const uint32_t num_iterations = div_ceil(
      mask_mode == MaskMode::kCausal
          ? min(kv_len, (bx + 1) * CTA_Q)
          : kv_len,
      CTA_K);

  int p = 1;
  for (uint32_t iter = 1; iter < num_iterations; iter++)
  { 
    // <NT> ��������ż����1=0^1, 0=1^1��ȷ����Ӧ���ݶ���ͬһbarrier�׶�
    p ^= 1;

    float dequant_scale = q_scale * K_scale[k_scale_idx + (iter - 1) * k_scale_advance_offset];
    sm_scale = original_sm_scale * dequant_scale;

    // wait for K
    wait(&barrier_K, p);

    // compute QK^T
    wgmma::warpgroup_arrive();
#pragma unroll
    // <NT> num_tiles_qΪ1����block�ĸ�ά�ȸպ���64������Ҫ���wgmma���з֡�
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
      int8_t *sQ_local = sQ + fq * 64 * head_dim;
      wgmma::wgmma_s8s8s32<CTA_K, 0, head_dim>(RS[fq], sQ_local, sK);
#pragma unroll
      // <NT> num_tiles_qk_inner��2��4����Ϊhead_dim��64��128��Q[CTA_Q, head_dim] * K[CTQ_K, head_dim]��
      // ������head_dim�䵱mma��k��wgmma_s8s8s32���õ���m64n128k32 �� m64n128k32�� kȡ32����Ҫ��k�����ֿ顣
      for (int k_it = 1; k_it < num_tiles_qk_inner; k_it++)
      {
        wgmma::wgmma_s8s8s32<CTA_K, 1, head_dim>(RS[fq], &sQ_local[k_it*32], &sK[k_it*32]);
      }
    }
    // <NT> �ύ��ͬ��wg
    wgmma::warpgroup_commit_batch();
    wgmma::warpgroup_wait<0>();

    // <NT> ���ּ����������ʼԤȡK��seq_len�������һ��Tile���ݡ�
    // load K
    if (threadIdx.x == 0)
    {
      expect_bytes<(CTA_K * head_dim) * sizeof(int8_t)>(&barrier_K);
      load_async_4D(sK, &tensorMapK, &barrier_K, 0, iter * CTA_K, kv_head_id, batch_id);
    }

    // <NT> RS��QK^T��ָ��ֿ������������ͨ��__int2float_rzתΪfp32��
    // ��ʼ����online softmax����Ļ��ڡ�RS_f32��[8,8], ��Ӧһ���̵߳����ݣ�
    // һ��wg��128���̣߳���Ӧһ��wgmma��m64n128k32�Ľ��[64,128].
    // convert RS to float
    float RS_f32[num_tiles_q][num_tiles_k][8];
#pragma unroll
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
#pragma unroll
      for (uint32_t fk = 0; fk < num_tiles_k; fk++)
      {
#pragma unroll
        for (uint32_t k = 0; k < 8; k++)
        {
          RS_f32[fq][fk][k] = __int2float_rz(RS[fq][fk][k]);
        }
      }
    }

    // <NT> ÿ���̻߳����µ�QKt���RS_f32[8,8]���Լ���ʷ�� m / d / RO, ��ͬ����online softmax��
    // ���� ���ֵ���£�ָ���͸��£�attention out���£�
    // RO: PV������ۼ�ֵ
    // m: ���ֵ����Ӧfa3�е�max_get_scale������row_max
    // d: ָ���ͣ���Ӧfa3�е�max_get_scale������row_sum
    update_mdo<num_tiles_q, num_tiles_k, num_tiles_v, false, true, false>(RS_f32, RO, m, d, sm_scale);

    // accumulate d on thread basis
#pragma unroll
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
#pragma unrol
      for (uint32_t fk = 0; fk < num_tiles_k; fk++)
      {
        d[fq][0] += (RS_f32[fq][fk][0] + RS_f32[fq][fk][1] + RS_f32[fq][fk][4] + RS_f32[fq][fk][5]);
        d[fq][1] += (RS_f32[fq][fk][2] + RS_f32[fq][fk][3] + RS_f32[fq][fk][6] + RS_f32[fq][fk][7]);
      }
    }

    // <NT> ��fp32��RSתΪfp8��׼������PV�ļ��㡣
    uint32_t RS_f8[num_tiles_q][num_tiles_pv_inner][4];
    RS_32_to_8<num_tiles_q, num_tiles_k>(RS_f32, RS_f8);

    // wait for V
    wait(&barrier_V, p);

    float RO_temp[num_tiles_q][num_tiles_v][8];
    wgmma::warpgroup_arrive();
#pragma unroll
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
      wgmma::wgmma_f8f8f32<head_dim, 0, CTA_K>(RO_temp[fq], RS_f8[fq][0], &sV[0]);
#pragma unroll
      for (uint32_t v_it = 1; v_it < num_tiles_pv_inner; v_it++)
      {
        wgmma::wgmma_f8f8f32<head_dim, 1, CTA_K>(RO_temp[fq], RS_f8[fq][v_it], &sV[v_it * 32]);
      }
    }

    wgmma::warpgroup_commit_batch();
    wgmma::warpgroup_wait<0>();

#pragma unroll
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
#pragma unroll
      for (uint32_t fv = 0; fv < num_tiles_v; fv++)
      {
#pragma unroll
        for (uint32_t k = 0; k < 8; k++)
        {
          RO[fq][fv][k] += RO_temp[fq][fv][k];
        }
      }
    }

    // <NT> ��һ��V�����꣬���Ͽ�ʼ��һ��V�Ķ�ȡ
    // load V
    if (threadIdx.x == 0)
    {
      expect_bytes<(CTA_K * head_dim) * sizeof(int8_t)>(&barrier_V);
      load_async_4D(sV, &tensorMapV, &barrier_V, iter * CTA_K, 0, kv_head_id, batch_id);
    }
  }

  { 
    p ^= 1;

    float dequant_scale = q_scale * K_scale[k_scale_idx + (num_iterations - 1) * k_scale_advance_offset];
    sm_scale = original_sm_scale;

    // wait for K
    wait(&barrier_K, p);

    // compute QK^T
    wgmma::warpgroup_arrive();
#pragma unroll
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
      int8_t *sQ_local = sQ + fq * 64 * head_dim;
      wgmma::wgmma_s8s8s32<CTA_K, 0, head_dim>(RS[fq], sQ_local, sK);
#pragma unroll
      for (int k_it = 1; k_it < num_tiles_qk_inner; k_it++)
      {
        wgmma::wgmma_s8s8s32<CTA_K, 1, head_dim>(RS[fq], &sQ_local[k_it*32], &sK[k_it*32]);
      }
    }
    wgmma::warpgroup_commit_batch();
    wgmma::warpgroup_wait<0>();

    // convert RS to float
    float RS_f32[num_tiles_q][num_tiles_k][8];
#pragma unroll
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
#pragma unroll
      for (uint32_t fk = 0; fk < num_tiles_k; fk++)
      {
#pragma unroll
        for (uint32_t k = 0; k < 8; k++)
        {
          RS_f32[fq][fk][k] = __int2float_rz(RS[fq][fk][k]) * dequant_scale;
        }
      }
    }

    // masking
#pragma unroll
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
#pragma unroll
      for (uint32_t fk = 0; fk < num_tiles_k; fk++)
      {
#pragma unroll
        for (uint32_t k = 0; k < 8; k++)
        {
          const uint32_t q_idx = Q_idx_lane_base + fq * 64 + 8 * ((k % 4) / 2);
          const uint32_t k_idx = (num_iterations - 1) * CTA_K + fk * 16 + 2 * (lane_id % 4) + 8 * (k / 4) + k % 2;

          bool is_out_of_bounds;

          if constexpr (mask_mode == MaskMode::kCausal)
          {
            is_out_of_bounds = (k_idx > q_idx) || (k_idx >= kv_len);
          }
          else
          {
            is_out_of_bounds = (k_idx >= kv_len);
          }

          if (is_out_of_bounds)
          {
            RS_f32[fq][fk][k] = -5000000.0f;
          }
        }
      }
    }

    update_mdo<num_tiles_q, num_tiles_k, num_tiles_v, false, true, false>(RS_f32, RO, m, d, sm_scale);

    // accumulate d on thread basis
#pragma unroll
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
#pragma unrol
      for (uint32_t fk = 0; fk < num_tiles_k; fk++)
      {
        d[fq][0] += (RS_f32[fq][fk][0] + RS_f32[fq][fk][1] + RS_f32[fq][fk][4] + RS_f32[fq][fk][5]);
        d[fq][1] += (RS_f32[fq][fk][2] + RS_f32[fq][fk][3] + RS_f32[fq][fk][6] + RS_f32[fq][fk][7]);
      }
    }

    uint32_t RS_f8[num_tiles_q][num_tiles_pv_inner][4];
    RS_32_to_8<num_tiles_q, num_tiles_k>(RS_f32, RS_f8);

    // wait for V
    wait(&barrier_V, p);

    float RO_temp[num_tiles_q][num_tiles_v][8];
    wgmma::warpgroup_arrive();
#pragma unroll
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
      wgmma::wgmma_f8f8f32<head_dim, 0, CTA_K>(RO_temp[fq], RS_f8[fq][0], &sV[0]);
#pragma unroll
      for (uint32_t v_it = 1; v_it < num_tiles_pv_inner; v_it++)
      {
        wgmma::wgmma_f8f8f32<head_dim, 1, CTA_K>(RO_temp[fq], RS_f8[fq][v_it], &sV[v_it * 32]);
      }
    }

    wgmma::warpgroup_commit_batch();
    wgmma::warpgroup_wait<0>();

#pragma unroll
    for (uint32_t fq = 0; fq < num_tiles_q; fq++)
    {
#pragma unroll
      for (uint32_t fv = 0; fv < num_tiles_v; fv++)
      {
#pragma unroll
        for (uint32_t k = 0; k < 8; k++)
        {
          RO[fq][fv][k] += RO_temp[fq][fv][k];
        }
      }
    }
  }

  normalize_d<num_tiles_q, num_tiles_v, ComputeUnit::kCudaCore>(RO, m, d);

  if constexpr (fuse_v_scale)
  {
    float v_scale[4];
    float *V_scale_base_ptr = V_scale +  batch_id * (num_qo_heads / num_kv_groups) * head_dim + (head_id / num_kv_groups) * head_dim + (lane_id % 4 ) * 2;
  #pragma unroll
    for (uint32_t fv = 0; fv < num_tiles_v; fv++)
    {
      ((float2*)v_scale)[0] = *((float2*)(V_scale_base_ptr + fv * 16));
      ((float2*)v_scale)[1] = *((float2*)(V_scale_base_ptr + fv * 16 + 8));

  #pragma unroll
      for (uint32_t fq = 0; fq < num_tiles_q; fq++)
      {
        RO[fq][fv][0] *= v_scale[0];
        RO[fq][fv][1] *= v_scale[1];
        RO[fq][fv][2] *= v_scale[0];
        RO[fq][fv][3] *= v_scale[1];
        RO[fq][fv][4] *= v_scale[2];
        RO[fq][fv][5] *= v_scale[3];
        RO[fq][fv][6] *= v_scale[2];
        RO[fq][fv][7] *= v_scale[3];
      }
    }
  }

  DTypeOut *O_lane_ptr = O + batch_id * stride_bz_o + head_id * stride_h_o + (bx * CTA_Q + warp_idx * 16 + (lane_id / 4)) * stride_seq_o + (lane_id % 4) * 2 ;
#pragma unroll
  for (uint32_t fq = 0; fq < num_tiles_q; fq++)
  {
#pragma unroll
    for (uint32_t fv = 0; fv < head_dim/16; fv++)
    { 
      if (Q_idx_lane_base + fq * 64 < qo_len)
      {
        if constexpr (std::is_same<DTypeOut, half>::value)
        {
          ((half2*)(O_lane_ptr + fq * 64 * stride_seq_o + fv * 16))[0] = __float22half2_rn(((float2*)(RO[fq][fv]))[0]);
          ((half2*)(O_lane_ptr + fq * 64 * stride_seq_o + fv * 16 + 8))[0] = __float22half2_rn(((float2*)(RO[fq][fv]))[2]);
        }
        else
        {
          ((nv_bfloat162*)(O_lane_ptr + fq * 64 * stride_seq_o + fv * 16))[0] = __float22bfloat162_rn(((float2*)(RO[fq][fv]))[0]);
          ((nv_bfloat162*)(O_lane_ptr + fq * 64 * stride_seq_o + fv * 16 + 8))[0] = __float22bfloat162_rn(((float2*)(RO[fq][fv]))[2]);  
        }
      }
      
      if (Q_idx_lane_base + fq * 64 + 8 < qo_len)
      {
        if constexpr (std::is_same<DTypeOut, half>::value)
        {
          ((half2*)(O_lane_ptr + fq * 64 * stride_seq_o + fv * 16 + 8 * stride_seq_o))[0] = __float22half2_rn(((float2*)(RO[fq][fv]))[1]);
          ((half2*)(O_lane_ptr + fq * 64 * stride_seq_o + fv * 16 + 8 + 8 * stride_seq_o))[0] = __float22half2_rn(((float2*)(RO[fq][fv]))[3]);
        }
        else
        {
          ((nv_bfloat162*)(O_lane_ptr + fq * 64 * stride_seq_o + fv * 16 + 8 * stride_seq_o))[0] = __float22bfloat162_rn(((float2*)(RO[fq][fv]))[1]);
          ((nv_bfloat162*)(O_lane_ptr + fq * 64 * stride_seq_o + fv * 16 + 8 + 8 * stride_seq_o))[0] = __float22bfloat162_rn(((float2*)(RO[fq][fv]))[3]);      
        }
      }
    }

    if constexpr (return_lse)
    {
      // only works for CTA_Q = 64
      uint32_t lse_idx = bx * CTA_Q + lane_id / 4 + 8 * (lane_id % 4) + 16 * warp_idx;
      float *lse_lane_ptr = Lse + batch_id * (qo_len * num_qo_heads) + head_id * qo_len + lse_idx;
      uint32_t fq = (lane_id % 4) / 2;
      uint32_t k = (lane_id % 4) % 2;

      if (lse_idx < qo_len && (lane_id % 4) < 2)
      {
        lse_lane_ptr[0] = (math::ptx_log2(d[fq][k]) + m[fq][k]);
      }
    }
  }
}

torch::Tensor qk_int8_sv_f8_accum_f32_attn_inst_buf(
                  torch::Tensor query,
                  torch::Tensor key,
                  torch::Tensor value,
                  torch::Tensor output,
                  torch::Tensor query_scale,
                  torch::Tensor key_scale,
                  int tensor_layout,
                  int is_causal,
                  int qk_quant_gran,
                  float sm_scale,
                  int return_lse)
{
  CHECK_CUDA(query);
  CHECK_CUDA(key);
  CHECK_CUDA(value);
  CHECK_CUDA(output);
  CHECK_CUDA(query_scale);
  CHECK_CUDA(key_scale);

  CHECK_LASTDIM_CONTIGUOUS(query);
  CHECK_LASTDIM_CONTIGUOUS(key);
  CHECK_LASTDIM_CONTIGUOUS(value);
  CHECK_LASTDIM_CONTIGUOUS(output);
  CHECK_CONTIGUOUS(query_scale);
  CHECK_CONTIGUOUS(key_scale);

  CHECK_DTYPE(query, torch::kInt8);
  CHECK_DTYPE(key, torch::kInt8);
  CHECK_DTYPE(value, at::ScalarType::Float8_e4m3fn);
  CHECK_DTYPE(query_scale, torch::kFloat32);
  CHECK_DTYPE(key_scale, torch::kFloat32);

  CHECK_DIMS(query, 4);
  CHECK_DIMS(key, 4);
  CHECK_DIMS(value, 4);
  CHECK_DIMS(output, 4);
  CHECK_DIMS(query_scale, 3);
  CHECK_DIMS(key_scale, 3);

  const int batch_size = query.size(0);
  const int head_dim = query.size(3);

  int stride_bz_q = query.stride(0);
  int stride_bz_k = key.stride(0);
  int stride_bz_v = value.stride(0);
  int stride_bz_o = output.stride(0);

  int qo_len, kv_len, padded_kv_len, num_qo_heads, num_kv_heads;
  int stride_seq_q, stride_h_q, stride_seq_k, stride_h_k, stride_h_v, stride_d_v, stride_seq_o, stride_h_o;

  assert(value.size(0) == batch_size);

  if (tensor_layout == 0)
  {
    qo_len = query.size(1);
    kv_len = key.size(1);
    num_qo_heads = query.size(2);
    num_kv_heads = key.size(2);

    stride_seq_q = query.stride(1);
    stride_h_q = query.stride(2);
    stride_seq_k = key.stride(1);
    stride_h_k = key.stride(2);
    stride_h_v = value.stride(2);
    stride_d_v = value.stride(1);
    stride_seq_o = output.stride(1);
    stride_h_o = output.stride(2);

    CHECK_SHAPE(key, batch_size, kv_len, num_kv_heads, head_dim);
    CHECK_SHAPE(output, batch_size, qo_len, num_qo_heads, head_dim);
    assert(value.size(1) == head_dim);
    assert(value.size(2) == num_kv_heads);
  }
  else
  {
    qo_len = query.size(2);
    kv_len = key.size(2);
    num_qo_heads = query.size(1);
    num_kv_heads = key.size(1);

    stride_seq_q = query.stride(2);
    stride_h_q = query.stride(1);
    stride_seq_k = key.stride(2);
    stride_h_k = key.stride(1);
    stride_h_v = value.stride(1);
    stride_d_v = value.stride(2);
    stride_seq_o = output.stride(2);
    stride_h_o = output.stride(1);

    CHECK_SHAPE(key, batch_size, num_kv_heads, kv_len, head_dim);
    CHECK_SHAPE(output, batch_size, num_qo_heads, qo_len, head_dim);
    assert(value.size(2) == head_dim);
    assert(value.size(1) == num_kv_heads);
  }

  if (num_qo_heads % num_kv_heads != 0) {
    std::ostringstream err_msg;
    err_msg << "num_qo_heads (" << num_qo_heads << ") must be divisible by num_kv_heads (" << num_kv_heads << ")";
    throw std::invalid_argument(err_msg.str());  
  }

  torch::Tensor lse = torch::empty({0});
  if (return_lse)
  {
    lse = torch::empty({batch_size, num_qo_heads, qo_len}, query.options().dtype(torch::kFloat32));
  }

  const int num_kv_groups = num_qo_heads / num_kv_heads;

  auto output_type = output.scalar_type();

  DISPATCH_HEAD_DIM(head_dim, HEAD_DIM, {
    DISPATCH_CAUSAL(is_causal, IS_CAUSAL, {
      DISPATCH_QK_QUANT_GRAN(qk_quant_gran, QK_QUANT_GRAN, {
        DISPATCH_RETURN_LSE(return_lse, RETURN_LSE, {
          DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(output_type, DTypeOut, {
            constexpr int CTA_Q = 64;
            constexpr int CTA_K = 128;
            constexpr int NUM_THREADS = 128;

            constexpr MaskMode mask_mode = IS_CAUSAL ? MaskMode::kCausal : MaskMode::kNone;

            assert(value.size(3) >= div_ceil(kv_len, CTA_K) * CTA_K);

            if constexpr (QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerWarp))
            {
              CHECK_SHAPE(query_scale, batch_size, num_qo_heads, static_cast<long>(div_ceil(qo_len, CTA_Q) * (NUM_THREADS / 32)));
              CHECK_SHAPE(key_scale, batch_size, num_kv_heads, static_cast<long>(div_ceil(kv_len, CTA_K)));
            }
            else if constexpr (QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerThread))
            {
              CHECK_SHAPE(query_scale, batch_size, num_qo_heads, static_cast<long>(div_ceil(qo_len, CTA_Q) * (NUM_THREADS / 32) * 8));
              CHECK_SHAPE(key_scale, batch_size, num_kv_heads, static_cast<long>(div_ceil(kv_len, CTA_K) * 4));    
            }
            else
            {
              static_assert(QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerWarp) || QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerThread), "Unsupported quantization granularity");
            }

            CUtensorMap tma_map_Q = create_tensor_map_4D<CTA_Q, HEAD_DIM>(reinterpret_cast<int8_t*>(query.data_ptr()), batch_size, num_qo_heads, qo_len, HEAD_DIM, stride_bz_q, stride_h_q, stride_seq_q);
            CUtensorMap tma_map_K = create_tensor_map_4D<CTA_K, HEAD_DIM>(reinterpret_cast<int8_t*>(key.data_ptr()), batch_size, num_kv_heads, kv_len, HEAD_DIM, stride_bz_k, stride_h_k, stride_seq_k);
            CUtensorMap tma_map_V = create_tensor_map_4D<HEAD_DIM, CTA_K>(reinterpret_cast<int8_t*>(value.data_ptr()), batch_size, num_kv_heads, HEAD_DIM, value.size(3), stride_bz_v, stride_h_v, stride_d_v);

            auto* kernel = qk_int8_sv_f8_attn_kernel<CTA_Q, CTA_K, NUM_THREADS, HEAD_DIM, static_cast<QuantGranularity>(QK_QUANT_GRAN), static_cast<QuantGranularity>(QK_QUANT_GRAN), DTypeOut, mask_mode, RETURN_LSE, false>;
            size_t sMemSize = CTA_Q * HEAD_DIM * sizeof(int8_t) + CTA_K * HEAD_DIM * sizeof(int8_t) + CTA_K * HEAD_DIM * sizeof(int8_t);
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                kernel),
                hipFuncAttributeMaxDynamicSharedMemorySize, sMemSize);
            
            dim3 grid(div_ceil(qo_len, CTA_Q), num_qo_heads, batch_size);
            kernel<<<grid, NUM_THREADS, sMemSize>>>(
              tma_map_Q,
              tma_map_K,
              tma_map_V,
              reinterpret_cast<float*>(query_scale.data_ptr()),
              reinterpret_cast<float*>(key_scale.data_ptr()),
              nullptr,
              reinterpret_cast<DTypeOut*>(output.data_ptr()),
              (RETURN_LSE) ? reinterpret_cast<float*>(lse.data_ptr()) : nullptr,
              stride_bz_o, stride_h_o, stride_seq_o,
              qo_len, kv_len, num_kv_groups, sm_scale);
          });
        });
      });
    });
  });

  return lse;
}

torch::Tensor qk_int8_sv_f8_accum_f32_fuse_v_scale_attn_inst_buf(
                    torch::Tensor query,
                    torch::Tensor key,
                    torch::Tensor value,
                    torch::Tensor output,
                    torch::Tensor query_scale,
                    torch::Tensor key_scale,
                    torch::Tensor value_scale,
                    int tensor_layout,
                    int is_causal,
                    int qk_quant_gran,
                    float sm_scale,
                    int return_lse)
{
  CHECK_CUDA(query);
  CHECK_CUDA(key);
  CHECK_CUDA(value);
  CHECK_CUDA(output);
  CHECK_CUDA(query_scale);
  CHECK_CUDA(key_scale);
  CHECK_CUDA(value_scale);

  CHECK_LASTDIM_CONTIGUOUS(query);
  CHECK_LASTDIM_CONTIGUOUS(key);
  CHECK_LASTDIM_CONTIGUOUS(value);
  CHECK_LASTDIM_CONTIGUOUS(output);
  CHECK_CONTIGUOUS(query_scale);
  CHECK_CONTIGUOUS(key_scale);
  CHECK_CONTIGUOUS(value_scale);

  CHECK_DTYPE(query, torch::kInt8);
  CHECK_DTYPE(key, torch::kInt8);
  CHECK_DTYPE(value, at::ScalarType::Float8_e4m3fn);
  CHECK_DTYPE(query_scale, torch::kFloat32);
  CHECK_DTYPE(key_scale, torch::kFloat32);
  CHECK_DTYPE(value_scale, torch::kFloat32);

  CHECK_DIMS(query, 4);
  CHECK_DIMS(key, 4);
  CHECK_DIMS(value, 4);
  CHECK_DIMS(output, 4);
  CHECK_DIMS(query_scale, 3);
  CHECK_DIMS(key_scale, 3);
  CHECK_DIMS(value_scale, 3);

  const int batch_size = query.size(0);
  const int head_dim = query.size(3);

  int stride_bz_q = query.stride(0);
  int stride_bz_k = key.stride(0);
  int stride_bz_v = value.stride(0);
  int stride_bz_o = output.stride(0);

  int qo_len, kv_len, padded_kv_len, num_qo_heads, num_kv_heads;
  int stride_seq_q, stride_h_q, stride_seq_k, stride_h_k, stride_h_v, stride_d_v, stride_seq_o, stride_h_o;

  assert(value.size(0) == batch_size);

  if (tensor_layout == 0)
  {
    qo_len = query.size(1);
    kv_len = key.size(1);
    num_qo_heads = query.size(2);
    num_kv_heads = key.size(2);

    stride_seq_q = query.stride(1);
    stride_h_q = query.stride(2);
    stride_seq_k = key.stride(1);
    stride_h_k = key.stride(2);
    stride_h_v = value.stride(2);
    stride_d_v = value.stride(1);
    stride_seq_o = output.stride(1);
    stride_h_o = output.stride(2);

    CHECK_SHAPE(key, batch_size, kv_len, num_kv_heads, head_dim);
    CHECK_SHAPE(output, batch_size, qo_len, num_qo_heads, head_dim);
    assert(value.size(1) == head_dim);
    assert(value.size(2) == num_kv_heads);
  }
  else
  {
    qo_len = query.size(2);
    kv_len = key.size(2);
    num_qo_heads = query.size(1);
    num_kv_heads = key.size(1);

    stride_seq_q = query.stride(2);
    stride_h_q = query.stride(1);
    stride_seq_k = key.stride(2);
    stride_h_k = key.stride(1);
    stride_h_v = value.stride(1);
    stride_d_v = value.stride(2);
    stride_seq_o = output.stride(2);
    stride_h_o = output.stride(1);

    CHECK_SHAPE(key, batch_size, num_kv_heads, kv_len, head_dim);
    CHECK_SHAPE(output, batch_size, num_qo_heads, qo_len, head_dim);
    assert(value.size(2) == head_dim);
    assert(value.size(1) == num_kv_heads);
  }

  if (num_qo_heads % num_kv_heads != 0) {
    std::ostringstream err_msg;
    err_msg << "num_qo_heads (" << num_qo_heads << ") must be divisible by num_kv_heads (" << num_kv_heads << ")";
    throw std::invalid_argument(err_msg.str());  
  }

  torch::Tensor lse = torch::empty({0});
  if (return_lse)
  {
    lse = torch::empty({batch_size, num_qo_heads, qo_len}, query.options().dtype(torch::kFloat32));
  }

  const int num_kv_groups = num_qo_heads / num_kv_heads;

  auto output_dtype = output.scalar_type();

  DISPATCH_HEAD_DIM(head_dim, HEAD_DIM, {
    DISPATCH_CAUSAL(is_causal, IS_CAUSAL, {
      DISPATCH_QK_QUANT_GRAN(qk_quant_gran, QK_QUANT_GRAN, {
        DISPATCH_RETURN_LSE(return_lse, RETURN_LSE, {
          DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(output_dtype, DTypeOut, {
            constexpr int CTA_Q = 64;
            constexpr int CTA_K = 128;
            constexpr int NUM_THREADS = 128;

            constexpr MaskMode mask_mode = IS_CAUSAL ? MaskMode::kCausal : MaskMode::kNone;

            assert(value.size(3) >= div_ceil(kv_len, CTA_K) * CTA_K);

            if constexpr (QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerWarp))
            {
              CHECK_SHAPE(query_scale, batch_size, num_qo_heads, static_cast<long>(div_ceil(qo_len, CTA_Q) * (NUM_THREADS / 32)));
              CHECK_SHAPE(key_scale, batch_size, num_kv_heads, static_cast<long>(div_ceil(kv_len, CTA_K)));
            }
            else if constexpr (QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerThread))
            {
              CHECK_SHAPE(query_scale, batch_size, num_qo_heads, static_cast<long>(div_ceil(qo_len, CTA_Q) * (NUM_THREADS / 32) * 8));
              CHECK_SHAPE(key_scale, batch_size, num_kv_heads, static_cast<long>(div_ceil(kv_len, CTA_K) * 4));    
            }
            else
            {
              static_assert(QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerWarp) || QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerThread), "Unsupported quantization granularity");
            }

            CHECK_SHAPE(value_scale, batch_size, num_kv_heads, head_dim);

            CUtensorMap tma_map_Q = create_tensor_map_4D<CTA_Q, HEAD_DIM>(reinterpret_cast<int8_t*>(query.data_ptr()), batch_size, num_qo_heads, qo_len, HEAD_DIM, stride_bz_q, stride_h_q, stride_seq_q);
            CUtensorMap tma_map_K = create_tensor_map_4D<CTA_K, HEAD_DIM>(reinterpret_cast<int8_t*>(key.data_ptr()), batch_size, num_kv_heads, kv_len, HEAD_DIM, stride_bz_k, stride_h_k, stride_seq_k);
            CUtensorMap tma_map_V = create_tensor_map_4D<HEAD_DIM, CTA_K>(reinterpret_cast<int8_t*>(value.data_ptr()), batch_size, num_kv_heads, HEAD_DIM, value.size(3), stride_bz_v, stride_h_v, stride_d_v);

            auto* kernel = qk_int8_sv_f8_attn_kernel<CTA_Q, CTA_K, NUM_THREADS, HEAD_DIM,  static_cast<QuantGranularity>(QK_QUANT_GRAN), static_cast<QuantGranularity>(QK_QUANT_GRAN), DTypeOut, mask_mode, RETURN_LSE, true>;
            size_t sMemSize = CTA_Q * HEAD_DIM * sizeof(int8_t) + CTA_K * HEAD_DIM * sizeof(int8_t) + CTA_K * HEAD_DIM * sizeof(int8_t);
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                kernel),
                hipFuncAttributeMaxDynamicSharedMemorySize, sMemSize);
            
            dim3 grid(div_ceil(qo_len, CTA_Q), num_qo_heads, batch_size);
            kernel<<<grid, NUM_THREADS, sMemSize>>>(
              tma_map_Q,
              tma_map_K,
              tma_map_V,
              reinterpret_cast<float*>(query_scale.data_ptr()),
              reinterpret_cast<float*>(key_scale.data_ptr()),
              reinterpret_cast<float*>(value_scale.data_ptr()),
              reinterpret_cast<DTypeOut*>(output.data_ptr()),
              (RETURN_LSE) ? reinterpret_cast<float*>(lse.data_ptr()) : nullptr,
              stride_bz_o, stride_h_o, stride_seq_o,
              qo_len, kv_len, num_kv_groups, sm_scale);
          });
        });
      });
    });
  });

  return lse;
}